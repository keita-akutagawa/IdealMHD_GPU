#include "hip/hip_runtime.h"
#include "boundary.hpp"


__global__
void periodicBoundaryX2nd_kernel(ConservationParameter* U)
{
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    if (j < device_ny) {
        U[j + 0 * device_ny].rho  = U[j + (device_nx-6) * device_ny].rho;
        U[j + 0 * device_ny].rhoU = U[j + (device_nx-6) * device_ny].rhoU;
        U[j + 0 * device_ny].rhoV = U[j + (device_nx-6) * device_ny].rhoV;
        U[j + 0 * device_ny].rhoW = U[j + (device_nx-6) * device_ny].rhoW;
        U[j + 0 * device_ny].bX   = U[j + (device_nx-6) * device_ny].bX;
        U[j + 0 * device_ny].bY   = U[j + (device_nx-6) * device_ny].bY;
        U[j + 0 * device_ny].bZ   = U[j + (device_nx-6) * device_ny].bZ;
        U[j + 0 * device_ny].e    = U[j + (device_nx-6) * device_ny].e;
        U[j + 1 * device_ny].rho  = U[j + (device_nx-5) * device_ny].rho;
        U[j + 1 * device_ny].rhoU = U[j + (device_nx-5) * device_ny].rhoU;
        U[j + 1 * device_ny].rhoV = U[j + (device_nx-5) * device_ny].rhoV;
        U[j + 1 * device_ny].rhoW = U[j + (device_nx-5) * device_ny].rhoW;
        U[j + 1 * device_ny].bX   = U[j + (device_nx-5) * device_ny].bX;
        U[j + 1 * device_ny].bY   = U[j + (device_nx-5) * device_ny].bY;
        U[j + 1 * device_ny].bZ   = U[j + (device_nx-5) * device_ny].bZ;
        U[j + 1 * device_ny].e    = U[j + (device_nx-5) * device_ny].e;
        U[j + 2 * device_ny].rho  = U[j + (device_nx-4) * device_ny].rho;
        U[j + 2 * device_ny].rhoU = U[j + (device_nx-4) * device_ny].rhoU;
        U[j + 2 * device_ny].rhoV = U[j + (device_nx-4) * device_ny].rhoV;
        U[j + 2 * device_ny].rhoW = U[j + (device_nx-4) * device_ny].rhoW;
        U[j + 2 * device_ny].bX   = U[j + (device_nx-4) * device_ny].bX;
        U[j + 2 * device_ny].bY   = U[j + (device_nx-4) * device_ny].bY;
        U[j + 2 * device_ny].bZ   = U[j + (device_nx-4) * device_ny].bZ;
        U[j + 2 * device_ny].e    = U[j + (device_nx-4) * device_ny].e;

        U[j + (device_nx-3) * device_ny].rho  = U[j + 3 * device_ny].rho;
        U[j + (device_nx-3) * device_ny].rhoU = U[j + 3 * device_ny].rhoU;
        U[j + (device_nx-3) * device_ny].rhoV = U[j + 3 * device_ny].rhoV;
        U[j + (device_nx-3) * device_ny].rhoW = U[j + 3 * device_ny].rhoW;
        U[j + (device_nx-3) * device_ny].bX   = U[j + 3 * device_ny].bX;
        U[j + (device_nx-3) * device_ny].bY   = U[j + 3 * device_ny].bY;
        U[j + (device_nx-3) * device_ny].bZ   = U[j + 3 * device_ny].bZ;
        U[j + (device_nx-3) * device_ny].e    = U[j + 3 * device_ny].e;
        U[j + (device_nx-2) * device_ny].rho  = U[j + 4 * device_ny].rho;
        U[j + (device_nx-2) * device_ny].rhoU = U[j + 4 * device_ny].rhoU;
        U[j + (device_nx-2) * device_ny].rhoV = U[j + 4 * device_ny].rhoV;
        U[j + (device_nx-2) * device_ny].rhoW = U[j + 4 * device_ny].rhoW;
        U[j + (device_nx-2) * device_ny].bX   = U[j + 4 * device_ny].bX;
        U[j + (device_nx-2) * device_ny].bY   = U[j + 4 * device_ny].bY;
        U[j + (device_nx-2) * device_ny].bZ   = U[j + 4 * device_ny].bZ;
        U[j + (device_nx-2) * device_ny].e    = U[j + 4 * device_ny].e;
        U[j + (device_nx-1) * device_ny].rho  = U[j + 5 * device_ny].rho;
        U[j + (device_nx-1) * device_ny].rhoU = U[j + 5 * device_ny].rhoU;
        U[j + (device_nx-1) * device_ny].rhoV = U[j + 5 * device_ny].rhoV;
        U[j + (device_nx-1) * device_ny].rhoW = U[j + 5 * device_ny].rhoW;
        U[j + (device_nx-1) * device_ny].bX   = U[j + 5 * device_ny].bX;
        U[j + (device_nx-1) * device_ny].bY   = U[j + 5 * device_ny].bY;
        U[j + (device_nx-1) * device_ny].bZ   = U[j + 5 * device_ny].bZ;
        U[j + (device_nx-1) * device_ny].e    = U[j + 5 * device_ny].e;
    }
}

void Boundary::periodicBoundaryX2nd(
    thrust::device_vector<ConservationParameter>& U
)
{
    int threadsPerBlock = 256;
    int blocksPerGrid = (ny + threadsPerBlock - 1) / threadsPerBlock;

    periodicBoundaryX2nd_kernel<<<blocksPerGrid, threadsPerBlock>>>(thrust::raw_pointer_cast(U.data()));

    hipDeviceSynchronize();
}


__global__
void symmetricBoundaryY2nd_kernel(ConservationParameter* U)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < device_nx) {
        U[0 + i * device_ny].rho  = U[5 + i * device_ny].rho;
        U[0 + i * device_ny].rhoU = U[5 + i * device_ny].rhoU;
        U[0 + i * device_ny].rhoV = U[5 + i * device_ny].rhoV;
        U[0 + i * device_ny].rhoW = U[5 + i * device_ny].rhoW;
        U[0 + i * device_ny].bX   = U[5 + i * device_ny].bX;
        U[0 + i * device_ny].bZ   = U[5 + i * device_ny].bZ;
        U[0 + i * device_ny].e    = U[5 + i * device_ny].e;
        U[1 + i * device_ny].rho  = U[4 + i * device_ny].rho;
        U[1 + i * device_ny].rhoU = U[4 + i * device_ny].rhoU;
        U[1 + i * device_ny].rhoV = U[4 + i * device_ny].rhoV;
        U[1 + i * device_ny].rhoW = U[4 + i * device_ny].rhoW;
        U[1 + i * device_ny].bX   = U[4 + i * device_ny].bX;
        U[1 + i * device_ny].bZ   = U[4 + i * device_ny].bZ;
        U[1 + i * device_ny].e    = U[4 + i * device_ny].e;
        U[2 + i * device_ny].rho  = U[3 + i * device_ny].rho;
        U[2 + i * device_ny].rhoU = U[3 + i * device_ny].rhoU;
        U[2 + i * device_ny].rhoV = U[3 + i * device_ny].rhoV;
        U[2 + i * device_ny].rhoW = U[3 + i * device_ny].rhoW;
        U[2 + i * device_ny].bX   = U[3 + i * device_ny].bX;
        U[2 + i * device_ny].bZ   = U[3 + i * device_ny].bZ;
        U[2 + i * device_ny].e    = U[3 + i * device_ny].e;

        U[0 + i * device_ny].bY   = U[4 + i * device_ny].bY;
        U[1 + i * device_ny].bY   = U[3 + i * device_ny].bY;

        U[device_ny-1 + i * device_ny].rho  = U[device_ny-6 + i * device_ny].rho;
        U[device_ny-1 + i * device_ny].rhoU = U[device_ny-6 + i * device_ny].rhoU;
        U[device_ny-1 + i * device_ny].rhoV = U[device_ny-6 + i * device_ny].rhoV;
        U[device_ny-1 + i * device_ny].rhoW = U[device_ny-6 + i * device_ny].rhoW;
        U[device_ny-1 + i * device_ny].bX   = U[device_ny-6 + i * device_ny].bX;
        U[device_ny-1 + i * device_ny].bZ   = U[device_ny-6 + i * device_ny].bZ;
        U[device_ny-1 + i * device_ny].e    = U[device_ny-6 + i * device_ny].e;
        U[device_ny-2 + i * device_ny].rho  = U[device_ny-5 + i * device_ny].rho;
        U[device_ny-2 + i * device_ny].rhoU = U[device_ny-5 + i * device_ny].rhoU;
        U[device_ny-2 + i * device_ny].rhoV = U[device_ny-5 + i * device_ny].rhoV;
        U[device_ny-2 + i * device_ny].rhoW = U[device_ny-5 + i * device_ny].rhoW;
        U[device_ny-2 + i * device_ny].bX   = U[device_ny-5 + i * device_ny].bX;
        U[device_ny-2 + i * device_ny].bZ   = U[device_ny-5 + i * device_ny].bZ;
        U[device_ny-2 + i * device_ny].e    = U[device_ny-5 + i * device_ny].e;
        U[device_ny-3 + i * device_ny].rho  = U[device_ny-4 + i * device_ny].rho;
        U[device_ny-3 + i * device_ny].rhoU = U[device_ny-4 + i * device_ny].rhoU;
        U[device_ny-3 + i * device_ny].rhoV = U[device_ny-4 + i * device_ny].rhoV;
        U[device_ny-3 + i * device_ny].rhoW = U[device_ny-4 + i * device_ny].rhoW;
        U[device_ny-3 + i * device_ny].bX   = U[device_ny-4 + i * device_ny].bX;
        U[device_ny-3 + i * device_ny].bZ   = U[device_ny-4 + i * device_ny].bZ;
        U[device_ny-3 + i * device_ny].e    = U[device_ny-4 + i * device_ny].e;

        U[device_ny-1 + i * device_ny].bY   = U[device_ny-5 + i * device_ny].bY;
        U[device_ny-2 + i * device_ny].bY   = U[device_ny-4 + i * device_ny].bY;
    }
}


void Boundary::symmetricBoundaryY2nd(
    thrust::device_vector<ConservationParameter>& U
)
{
    int threadsPerBlock = 256;
    int blocksPerGrid = (nx + threadsPerBlock - 1) / threadsPerBlock;

    symmetricBoundaryY2nd_kernel<<<blocksPerGrid, threadsPerBlock>>>(thrust::raw_pointer_cast(U.data()));

    hipDeviceSynchronize();
}

