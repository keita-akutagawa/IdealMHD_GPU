#include "hip/hip_runtime.h"
#include "boundary.hpp"


__global__
void periodicBoundaryX2nd_kernel(ConservationParameter* U)
{
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    U[j + 0 * device_ny].rho  = U[j + (device_nx-4) * device_ny].rho;
    U[j + 0 * device_ny].rhoU = U[j + (device_nx-4) * device_ny].rhoU;
    U[j + 0 * device_ny].rhoV = U[j + (device_nx-4) * device_ny].rhoV;
    U[j + 0 * device_ny].rhoW = U[j + (device_nx-4) * device_ny].rhoW;
    U[j + 0 * device_ny].bX   = U[j + (device_nx-4) * device_ny].bX;
    U[j + 0 * device_ny].bY   = U[j + (device_nx-4) * device_ny].bY;
    U[j + 0 * device_ny].bZ   = U[j + (device_nx-4) * device_ny].bZ;
    U[j + 0 * device_ny].e    = U[j + (device_nx-4) * device_ny].e;
    U[j + 1 * device_ny].rho  = U[j + (device_nx-3) * device_ny].rho;
    U[j + 1 * device_ny].rhoU = U[j + (device_nx-3) * device_ny].rhoU;
    U[j + 1 * device_ny].rhoV = U[j + (device_nx-3) * device_ny].rhoV;
    U[j + 1 * device_ny].rhoW = U[j + (device_nx-3) * device_ny].rhoW;
    U[j + 1 * device_ny].bX   = U[j + (device_nx-3) * device_ny].bX;
    U[j + 1 * device_ny].bY   = U[j + (device_nx-3) * device_ny].bY;
    U[j + 1 * device_ny].bZ   = U[j + (device_nx-3) * device_ny].bZ;
    U[j + 1 * device_ny].e    = U[j + (device_nx-3) * device_ny].e;

    U[j + (device_nx-2) * device_ny].rho  = U[j + 2 * device_ny].rho;
    U[j + (device_nx-2) * device_ny].rhoU = U[j + 2 * device_ny].rhoU;
    U[j + (device_nx-2) * device_ny].rhoV = U[j + 2 * device_ny].rhoV;
    U[j + (device_nx-2) * device_ny].rhoW = U[j + 2 * device_ny].rhoW;
    U[j + (device_nx-2) * device_ny].bX   = U[j + 2 * device_ny].bX;
    U[j + (device_nx-2) * device_ny].bY   = U[j + 2 * device_ny].bY;
    U[j + (device_nx-2) * device_ny].bZ   = U[j + 2 * device_ny].bZ;
    U[j + (device_nx-2) * device_ny].e    = U[j + 2 * device_ny].e;
    U[j + (device_nx-1) * device_ny].rho  = U[j + 3 * device_ny].rho;
    U[j + (device_nx-1) * device_ny].rhoU = U[j + 3 * device_ny].rhoU;
    U[j + (device_nx-1) * device_ny].rhoV = U[j + 3 * device_ny].rhoV;
    U[j + (device_nx-1) * device_ny].rhoW = U[j + 3 * device_ny].rhoW;
    U[j + (device_nx-1) * device_ny].bX   = U[j + 3 * device_ny].bX;
    U[j + (device_nx-1) * device_ny].bY   = U[j + 3 * device_ny].bY;
    U[j + (device_nx-1) * device_ny].bZ   = U[j + 3 * device_ny].bZ;
    U[j + (device_nx-1) * device_ny].e    = U[j + 3 * device_ny].e;
}

void Boundary::periodicBoundaryX2nd(
    thrust::device_vector<ConservationParameter>& U
)
{
    int threadsPerBlock = ny;
    int blocksPerGrid = 1;

    periodicBoundaryX2nd_kernel<<<blocksPerGrid, threadsPerBlock>>>(thrust::raw_pointer_cast(U.data()));

    hipDeviceSynchronize();
}


__global__
void periodicBoundaryY2nd_kernel(ConservationParameter* U)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    U[0 + i * device_ny].rho  = U[device_ny-4 + i * device_ny].rho;
    U[0 + i * device_ny].rhoU = U[device_ny-4 + i * device_ny].rhoU;
    U[0 + i * device_ny].rhoV = U[device_ny-4 + i * device_ny].rhoV;
    U[0 + i * device_ny].rhoW = U[device_ny-4 + i * device_ny].rhoW;
    U[0 + i * device_ny].bX   = U[device_ny-4 + i * device_ny].bX;
    U[0 + i * device_ny].bY   = U[device_ny-4 + i * device_ny].bY;
    U[0 + i * device_ny].bZ   = U[device_ny-4 + i * device_ny].bZ;
    U[0 + i * device_ny].e    = U[device_ny-4 + i * device_ny].e;
    U[1 + i * device_ny].rho  = U[device_ny-3 + i * device_ny].rho;
    U[1 + i * device_ny].rhoU = U[device_ny-3 + i * device_ny].rhoU;
    U[1 + i * device_ny].rhoV = U[device_ny-3 + i * device_ny].rhoV;
    U[1 + i * device_ny].rhoW = U[device_ny-3 + i * device_ny].rhoW;
    U[1 + i * device_ny].bX   = U[device_ny-3 + i * device_ny].bX;
    U[1 + i * device_ny].bY   = U[device_ny-3 + i * device_ny].bY;
    U[1 + i * device_ny].bZ   = U[device_ny-3 + i * device_ny].bZ;
    U[1 + i * device_ny].e    = U[device_ny-3 + i * device_ny].e;

    U[device_ny-2 + i * device_ny].rho  = U[2 + i * device_ny].rho;
    U[device_ny-2 + i * device_ny].rhoU = U[2 + i * device_ny].rhoU;
    U[device_ny-2 + i * device_ny].rhoV = U[2 + i * device_ny].rhoV;
    U[device_ny-2 + i * device_ny].rhoW = U[2 + i * device_ny].rhoW;
    U[device_ny-2 + i * device_ny].bX   = U[2 + i * device_ny].bX;
    U[device_ny-2 + i * device_ny].bY   = U[2 + i * device_ny].bY;
    U[device_ny-2 + i * device_ny].bZ   = U[2 + i * device_ny].bZ;
    U[device_ny-2 + i * device_ny].e    = U[2 + i * device_ny].e;
    U[device_ny-1 + i * device_ny].rho  = U[3 + i * device_ny].rho;
    U[device_ny-1 + i * device_ny].rhoU = U[3 + i * device_ny].rhoU;
    U[device_ny-1 + i * device_ny].rhoV = U[3 + i * device_ny].rhoV;
    U[device_ny-1 + i * device_ny].rhoW = U[3 + i * device_ny].rhoW;
    U[device_ny-1 + i * device_ny].bX   = U[3 + i * device_ny].bX;
    U[device_ny-1 + i * device_ny].bY   = U[3 + i * device_ny].bY;
    U[device_ny-1 + i * device_ny].bZ   = U[3 + i * device_ny].bZ;
    U[device_ny-1 + i * device_ny].e    = U[3 + i * device_ny].e;
}

void Boundary::periodicBoundaryY2nd(
    thrust::device_vector<ConservationParameter>& U
)
{
    int threadsPerBlock = nx;
    int blocksPerGrid = 1;

    periodicBoundaryY2nd_kernel<<<blocksPerGrid, threadsPerBlock>>>(thrust::raw_pointer_cast(U.data()));

    hipDeviceSynchronize();
}

///////////////////////

__global__
void symmetricBoundaryX2nd_kernel(ConservationParameter* U)
{
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    U[j + 0 * device_ny].rho  = U[j + 2 * device_ny].rho;
    U[j + 0 * device_ny].rhoU = U[j + 2 * device_ny].rhoU;
    U[j + 0 * device_ny].rhoV = U[j + 2 * device_ny].rhoV;
    U[j + 0 * device_ny].rhoW = U[j + 2 * device_ny].rhoW;
    U[j + 0 * device_ny].bX   = U[j + 2 * device_ny].bX;
    U[j + 0 * device_ny].bY   = U[j + 2 * device_ny].bY;
    U[j + 0 * device_ny].bZ   = U[j + 2 * device_ny].bZ;
    U[j + 0 * device_ny].e    = U[j + 2 * device_ny].e;
    U[j + 1 * device_ny].rho  = U[j + 2 * device_ny].rho;
    U[j + 1 * device_ny].rhoU = U[j + 2 * device_ny].rhoU;
    U[j + 1 * device_ny].rhoV = U[j + 2 * device_ny].rhoV;
    U[j + 1 * device_ny].rhoW = U[j + 2 * device_ny].rhoW;
    U[j + 1 * device_ny].bX   = U[j + 2 * device_ny].bX;
    U[j + 1 * device_ny].bY   = U[j + 2 * device_ny].bY;
    U[j + 1 * device_ny].bZ   = U[j + 2 * device_ny].bZ;
    U[j + 1 * device_ny].e    = U[j + 2 * device_ny].e;

    U[j + (device_nx-1) * device_ny].rho  = U[j + (device_nx-3) * device_ny].rho;
    U[j + (device_nx-1) * device_ny].rhoU = U[j + (device_nx-3) * device_ny].rhoU;
    U[j + (device_nx-1) * device_ny].rhoV = U[j + (device_nx-3) * device_ny].rhoV;
    U[j + (device_nx-1) * device_ny].rhoW = U[j + (device_nx-3) * device_ny].rhoW;
    U[j + (device_nx-1) * device_ny].bX   = U[j + (device_nx-3) * device_ny].bX;
    U[j + (device_nx-1) * device_ny].bY   = U[j + (device_nx-3) * device_ny].bY;
    U[j + (device_nx-1) * device_ny].bZ   = U[j + (device_nx-3) * device_ny].bZ;
    U[j + (device_nx-1) * device_ny].e    = U[j + (device_nx-3) * device_ny].e;
    U[j + (device_nx-2) * device_ny].rho  = U[j + (device_nx-3) * device_ny].rho;
    U[j + (device_nx-2) * device_ny].rhoU = U[j + (device_nx-3) * device_ny].rhoU;
    U[j + (device_nx-2) * device_ny].rhoV = U[j + (device_nx-3) * device_ny].rhoV;
    U[j + (device_nx-2) * device_ny].rhoW = U[j + (device_nx-3) * device_ny].rhoW;
    U[j + (device_nx-2) * device_ny].bX   = U[j + (device_nx-3) * device_ny].bX;
    U[j + (device_nx-2) * device_ny].bY   = U[j + (device_nx-3) * device_ny].bY;
    U[j + (device_nx-2) * device_ny].bZ   = U[j + (device_nx-3) * device_ny].bZ;
    U[j + (device_nx-2) * device_ny].e    = U[j + (device_nx-3) * device_ny].e;
}


void Boundary::symmetricBoundaryX2nd(
    thrust::device_vector<ConservationParameter>& U
)
{
    int threadsPerBlock = ny;
    int blocksPerGrid = 1;

    symmetricBoundaryX2nd_kernel<<<blocksPerGrid, threadsPerBlock>>>(thrust::raw_pointer_cast(U.data()));

    hipDeviceSynchronize();
}


__global__
void symmetricBoundaryY2nd_kernel(ConservationParameter* U)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    U[0 + i * device_ny].rho  = U[2 + i * device_ny].rho;
    U[0 + i * device_ny].rhoU = U[2 + i * device_ny].rhoU;
    U[0 + i * device_ny].rhoV = U[2 + i * device_ny].rhoV;
    U[0 + i * device_ny].rhoW = U[2 + i * device_ny].rhoW;
    U[0 + i * device_ny].bX   = U[2 + i * device_ny].bX;
    U[0 + i * device_ny].bY   = U[2 + i * device_ny].bY;
    U[0 + i * device_ny].bZ   = U[2 + i * device_ny].bZ;
    U[0 + i * device_ny].e    = U[2 + i * device_ny].e;
    U[1 + i * device_ny].rho  = U[2 + i * device_ny].rho;
    U[1 + i * device_ny].rhoU = U[2 + i * device_ny].rhoU;
    U[1 + i * device_ny].rhoV = U[2 + i * device_ny].rhoV;
    U[1 + i * device_ny].rhoW = U[2 + i * device_ny].rhoW;
    U[1 + i * device_ny].bX   = U[2 + i * device_ny].bX;
    U[1 + i * device_ny].bY   = U[2 + i * device_ny].bY;
    U[1 + i * device_ny].bZ   = U[2 + i * device_ny].bZ;
    U[1 + i * device_ny].e    = U[2 + i * device_ny].e;

    U[device_nx-1 + i * device_ny].rho  = U[device_nx-3 + i * device_ny].rho;
    U[device_nx-1 + i * device_ny].rhoU = U[device_nx-3 + i * device_ny].rhoU;
    U[device_nx-1 + i * device_ny].rhoV = U[device_nx-3 + i * device_ny].rhoV;
    U[device_nx-1 + i * device_ny].rhoW = U[device_nx-3 + i * device_ny].rhoW;
    U[device_nx-1 + i * device_ny].bX   = U[device_nx-3 + i * device_ny].bX;
    U[device_nx-1 + i * device_ny].bY   = U[device_nx-3 + i * device_ny].bY;
    U[device_nx-1 + i * device_ny].bZ   = U[device_nx-3 + i * device_ny].bZ;
    U[device_nx-1 + i * device_ny].e    = U[device_nx-3 + i * device_ny].e;
    U[device_nx-2 + i * device_ny].rho  = U[device_nx-3 + i * device_ny].rho;
    U[device_nx-2 + i * device_ny].rhoU = U[device_nx-3 + i * device_ny].rhoU;
    U[device_nx-2 + i * device_ny].rhoV = U[device_nx-3 + i * device_ny].rhoV;
    U[device_nx-2 + i * device_ny].rhoW = U[device_nx-3 + i * device_ny].rhoW;
    U[device_nx-2 + i * device_ny].bX   = U[device_nx-3 + i * device_ny].bX;
    U[device_nx-2 + i * device_ny].bY   = U[device_nx-3 + i * device_ny].bY;
    U[device_nx-2 + i * device_ny].bZ   = U[device_nx-3 + i * device_ny].bZ;
    U[device_nx-2 + i * device_ny].e    = U[device_nx-3 + i * device_ny].e;
}


void Boundary::symmetricBoundaryY2nd(
    thrust::device_vector<ConservationParameter>& U
)
{
    int threadsPerBlock = nx;
    int blocksPerGrid = 1;

    symmetricBoundaryY2nd_kernel<<<blocksPerGrid, threadsPerBlock>>>(thrust::raw_pointer_cast(U.data()));

    hipDeviceSynchronize();
}

