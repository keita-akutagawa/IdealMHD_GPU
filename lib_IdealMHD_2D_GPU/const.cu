#include "const.hpp"


void initializeDeviceConstants() {
    hipMemcpyToSymbol(HIP_SYMBOL(device_EPS), &EPS, sizeof(double));
    hipMemcpyToSymbol(HIP_SYMBOL(device_PI), &PI, sizeof(double));

    hipMemcpyToSymbol(HIP_SYMBOL(device_dx), &dx, sizeof(double));
    hipMemcpyToSymbol(HIP_SYMBOL(device_xmin), &xmin, sizeof(double));
    hipMemcpyToSymbol(HIP_SYMBOL(device_xmax), &xmax, sizeof(double));
    hipMemcpyToSymbol(HIP_SYMBOL(device_nx), &nx, sizeof(int));

    hipMemcpyToSymbol(HIP_SYMBOL(device_dy), &dy, sizeof(double));
    hipMemcpyToSymbol(HIP_SYMBOL(device_ymin), &ymin, sizeof(double));
    hipMemcpyToSymbol(HIP_SYMBOL(device_ymax), &ymax, sizeof(double));
    hipMemcpyToSymbol(HIP_SYMBOL(device_ny), &ny, sizeof(int));

    hipMemcpyToSymbol(HIP_SYMBOL(device_CFL), &CFL, sizeof(double));
    hipMemcpyToSymbol(HIP_SYMBOL(device_gamma_mhd), &gamma_mhd, sizeof(double));

    hipMemcpyToSymbol(HIP_SYMBOL(device_dt), &dt, sizeof(double));

    hipMemcpyToSymbol(HIP_SYMBOL(device_totalStep), &totalStep, sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(totalTime), &device_totalTime, sizeof(double));
}
