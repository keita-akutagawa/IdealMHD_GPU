#include "hip/hip_runtime.h"
#include "calculate_Q.hpp"


CalculateQ::CalculateQ(MPIInfo& mPIInfo)
    : mPIInfo(mPIInfo), 
      muscl(mPIInfo)
{
}

__global__ void getBasicParamter_kernel(
    const ConservationParameter* U, 
    BasicParameter* dQ, 
    int localSizeX, int localSizeY, int shiftForNeighbor
)
{

    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i < localSizeX - 1 && j < localSizeY - 1) {

        double rho, u, v, w, bX, bY, bZ, e, p, psi;
        int index = j + i * localSizeY;

        rho = U[index].rho;
        u   = U[index].rhoU / rho;
        v   = U[index].rhoV / rho;
        w   = U[index].rhoW / rho;
        bX  = U[index].bX;
        bY  = U[index].bY;
        bZ  = U[index].bZ;
        e   = U[index].e;
        p   = (device_gamma_mhd - 1.0)
            * (e - 0.5 * (rho * (u * u + v * v + w * w))
            - 0.5 * (bX * bX + bY * bY + bZ * bZ));
        psi = U[index].psi; 
        
        dQ[index].rho = rho;
        dQ[index].u   = u;
        dQ[index].v   = v;
        dQ[index].w   = w;
        dQ[index].bX  = bX; 
        dQ[index].bY  = bY;
        dQ[index].bZ  = bZ;
        dQ[index].p   = p;
        dQ[index].psi = psi; 
    }
}

void CalculateQ::setPhysicalParameterX(
    const thrust::device_vector<ConservationParameter>& U, 
    thrust::device_vector<BasicParameter>& dQCenter
)
{

    dim3 threadsPerBlock(16, 16);
    dim3 blocksPerGrid((mPIInfo.localSizeX + threadsPerBlock.x - 1) / threadsPerBlock.x,
                       (mPIInfo.localSizeY + threadsPerBlock.y - 1) / threadsPerBlock.y);
    
    getBasicParamter_kernel<<<blocksPerGrid, threadsPerBlock>>>(
        thrust::raw_pointer_cast(U.data()), 
        thrust::raw_pointer_cast(dQCenter.data()), 
        mPIInfo.localSizeX, mPIInfo.localSizeY, mPIInfo.localSizeY
    );
    hipDeviceSynchronize();
}

void CalculateQ::setPhysicalParameterY(
    const thrust::device_vector<ConservationParameter>& U, 
    thrust::device_vector<BasicParameter>& dQCenter
)
{
    dim3 threadsPerBlock(16, 16);
    dim3 blocksPerGrid((mPIInfo.localSizeX + threadsPerBlock.x - 1) / threadsPerBlock.x,
                       (mPIInfo.localSizeY + threadsPerBlock.y - 1) / threadsPerBlock.y);
    
    getBasicParamter_kernel<<<blocksPerGrid, threadsPerBlock>>>(
        thrust::raw_pointer_cast(U.data()), 
        thrust::raw_pointer_cast(dQCenter.data()), 
        mPIInfo.localSizeX, mPIInfo.localSizeY, 1
    );
    hipDeviceSynchronize();
}


void CalculateQ::calculateLeftQX(
    const thrust::device_vector<BasicParameter>& dQCenter, 
    thrust::device_vector<BasicParameter>& dQLeft
)
{ 
    muscl.getLeftQX(dQCenter, dQLeft);
}


void CalculateQ::calculateLeftQY(
    const thrust::device_vector<BasicParameter>& dQCenter, 
    thrust::device_vector<BasicParameter>& dQLeft
)
{ 
    muscl.getLeftQY(dQCenter, dQLeft);
}


void CalculateQ::calculateRightQX(
    const thrust::device_vector<BasicParameter>& dQCenter, 
    thrust::device_vector<BasicParameter>& dQRight
)
{ 
    muscl.getRightQX(dQCenter, dQRight);
}


void CalculateQ::calculateRightQY(
    const thrust::device_vector<BasicParameter>& dQCenter, 
    thrust::device_vector<BasicParameter>& dQRight
)
{ 
    muscl.getRightQY(dQCenter, dQRight);
}


