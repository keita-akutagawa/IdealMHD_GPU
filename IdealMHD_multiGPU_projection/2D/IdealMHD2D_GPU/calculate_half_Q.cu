#include "hip/hip_runtime.h"
#include "calculate_half_Q.hpp"


CalculateHalfQ::CalculateHalfQ(MPIInfo& mPIInfo)
    : mPIInfo(mPIInfo), 
      muscl(mPIInfo)
{
}


__global__ void getBasicParamter_kernel(
    const ConservationParameter* U, 
    BasicParameter* dQ, 
    int localSizeX, int localSizeY, int shiftForNeighbor
)
{

    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i < localSizeX - 1 && j < localSizeY - 1) {

        double rho, u, v, w, bX, bY, bZ, e, p;
        int index = j + i * localSizeY;

        rho     = U[index].rho;
        u       = U[index].rhoU / rho;
        v       = U[index].rhoV / rho;
        w       = U[index].rhoW / rho;
        bX      = 0.5 * (U[index].bX + U[index + shiftForNeighbor].bX); // flux計算でx, y方向使いまわすため
        bY      = U[index].bY;
        bZ      = U[index].bZ;
        e       = U[index].e;
        p       = (device_gamma_mhd - 1.0)
                * (e - 0.5 * (rho * (u * u + v * v + w * w))
                - 0.5 * (bX * bX + bY * bY + bZ * bZ));
        
        dQ[index].rho = rho;
        dQ[index].u   = u;
        dQ[index].v   = v;
        dQ[index].w   = w;
        dQ[index].bX  = bX; //HLLDではBxは中心のものを使うため
        dQ[index].bY  = bY;
        dQ[index].bZ  = bZ;
        dQ[index].p   = p;
    }
}


void CalculateHalfQ::setPhysicalParameterX(
    const thrust::device_vector<ConservationParameter>& U, 
    thrust::device_vector<BasicParameter>& dQCenter
)
{

    dim3 threadsPerBlock(16, 16);
    dim3 blocksPerGrid((mPIInfo.localSizeX + threadsPerBlock.x - 1) / threadsPerBlock.x,
                       (mPIInfo.localSizeY + threadsPerBlock.y - 1) / threadsPerBlock.y);
    
    getBasicParamter_kernel<<<blocksPerGrid, threadsPerBlock>>>(
        thrust::raw_pointer_cast(U.data()), 
        thrust::raw_pointer_cast(dQCenter.data()), 
        mPIInfo.localSizeX, mPIInfo.localSizeY, mPIInfo.localSizeY
    );
    hipDeviceSynchronize();
}

void CalculateHalfQ::setPhysicalParameterY(
    const thrust::device_vector<ConservationParameter>& U, 
    thrust::device_vector<BasicParameter>& dQCenter
)
{
    dim3 threadsPerBlock(16, 16);
    dim3 blocksPerGrid((mPIInfo.localSizeX + threadsPerBlock.x - 1) / threadsPerBlock.x,
                       (mPIInfo.localSizeY + threadsPerBlock.y - 1) / threadsPerBlock.y);
    
    getBasicParamter_kernel<<<blocksPerGrid, threadsPerBlock>>>(
        thrust::raw_pointer_cast(U.data()), 
        thrust::raw_pointer_cast(dQCenter.data()), 
        mPIInfo.localSizeX, mPIInfo.localSizeY, 1
    );
    hipDeviceSynchronize();
}


void CalculateHalfQ::calculateLeftQX(
    const thrust::device_vector<BasicParameter>& dQCenter, 
    thrust::device_vector<BasicParameter>& dQLeft
)
{ 
    muscl.getLeftQX(dQCenter, dQLeft);
}


void CalculateHalfQ::calculateLeftQY(
    const thrust::device_vector<BasicParameter>& dQCenter, 
    thrust::device_vector<BasicParameter>& dQLeft
)
{ 
    muscl.getLeftQY(dQCenter, dQLeft);
}


void CalculateHalfQ::calculateRightQX(
    const thrust::device_vector<BasicParameter>& dQCenter, 
    thrust::device_vector<BasicParameter>& dQRight
)
{ 
    muscl.getRightQX(dQCenter, dQRight);
}


void CalculateHalfQ::calculateRightQY(
    const thrust::device_vector<BasicParameter>& dQCenter, 
    thrust::device_vector<BasicParameter>& dQRight
)
{ 
    muscl.getRightQY(dQCenter, dQRight);
}
