#include "const.hpp"


void initializeDeviceConstants() {
    hipMemcpyToSymbol(HIP_SYMBOL(device_EPS), &EPS, sizeof(double));
    hipMemcpyToSymbol(HIP_SYMBOL(device_PI), &PI, sizeof(double));
    hipMemcpyToSymbol(HIP_SYMBOL(device_dx), &dx, sizeof(double));
    hipMemcpyToSymbol(HIP_SYMBOL(device_xmin), &xmin, sizeof(double));
    hipMemcpyToSymbol(HIP_SYMBOL(device_xmax), &xmax, sizeof(double));
    hipMemcpyToSymbol(HIP_SYMBOL(device_nx), &nx, sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(device_CFL), &CFL, sizeof(double));
    hipMemcpyToSymbol(HIP_SYMBOL(device_gamma_mhd), &gamma_mhd, sizeof(double));
    hipMemcpyToSymbol(HIP_SYMBOL(device_dt), &dt, sizeof(double));
    hipMemcpyToSymbol(HIP_SYMBOL(device_totalStep), &totalStep, sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(totalTime), &device_totalTime, sizeof(double));
}
