#include "hip/hip_runtime.h"
#include "boundary.hpp"
#include "const.hpp"


__global__
void periodicBoundary2nd_kernel(ConservationParameter* U)
{
    U[0].rho  = U[nx-4].rho;
    U[0].rhoU = U[nx-4].rhoU;
    U[0].rhoV = U[nx-4].rhoV;
    U[0].rhoW = U[nx-4].rhoW;
    U[0].bX   = U[nx-4].bX;
    U[0].bY   = U[nx-4].bY;
    U[0].bZ   = U[nx-4].bZ;
    U[0].e    = U[nx-4].e;
    U[1].rho  = U[nx-3].rho;
    U[1].rhoU = U[nx-3].rhoU;
    U[1].rhoV = U[nx-3].rhoV;
    U[1].rhoW = U[nx-3].rhoW;
    U[1].bX   = U[nx-3].bX;
    U[1].bY   = U[nx-3].bY;
    U[1].bZ   = U[nx-3].bZ;
    U[1].e    = U[nx-3].e;

    U[nx-2].rho  = U[2].rho;
    U[nx-2].rhoU = U[2].rhoU;
    U[nx-2].rhoV = U[2].rhoV;
    U[nx-2].rhoW = U[2].rhoW;
    U[nx-2].bX   = U[2].bX;
    U[nx-2].bY   = U[2].bY;
    U[nx-2].bZ   = U[2].bZ;
    U[nx-2].e    = U[2].e;
    U[nx-1].rho  = U[3].rho;
    U[nx-1].rhoU = U[3].rhoU;
    U[nx-1].rhoV = U[3].rhoV;
    U[nx-1].rhoW = U[3].rhoW;
    U[nx-1].bX   = U[3].bX;
    U[nx-1].bY   = U[3].bY;
    U[nx-1].bZ   = U[3].bZ;
    U[nx-1].e    = U[3].e;
}

void Boundary::periodicBoundary(
    thrust::device_vector<ConservationParameter>& U
)
{
    int threadsPerBlock = 1;
    int blocksPerGrid = 1;

    periodicBoundary2nd_kernel<<<blocksPerGrid, threadsPerBlock>>>(thrust::raw_pointer_cast(U.data()));

    hipDeviceSynchronize();
}


__global__
void symmetricBoundary2nd_kernel(ConservationParameter* U)
{
    
}


void Boundary::symmetricBoundary2nd(
    thrust::device_vector<ConservationParameter>& U
)
{
    int threadsPerBlock = 1;
    int blocksPerGrid = 1;

    symmetricBoundary2nd_kernel<<<blocksPerGrid, threadsPerBlock>>>(thrust::raw_pointer_cast(U.data()));

    hipDeviceSynchronize();
}

