#include "hip/hip_runtime.h"
#include "boundary.hpp"


Boundary::Boundary(MPIInfo& mPIInfo)
    : mPIInfo(mPIInfo)
{

    hipMalloc(&device_mPIInfo, sizeof(MPIInfo));
    hipMemcpy(device_mPIInfo, &mPIInfo, sizeof(MPIInfo), hipMemcpyHostToDevice);
    
}


void Boundary::periodicBoundaryX2nd_U(
    thrust::device_vector<ConservationParameter>& U
)
{

}


void Boundary::periodicBoundaryY2nd_U(
    thrust::device_vector<ConservationParameter>& U
)
{
    
}

void Boundary::periodicBoundaryX2nd_flux(
    thrust::device_vector<Flux>& fluxF, 
    thrust::device_vector<Flux>& fluxG
)
{

}


void Boundary::periodicBoundaryY2nd_flux(
    thrust::device_vector<Flux>& fluxF, 
    thrust::device_vector<Flux>& fluxG
)
{
    
}

///////////////////////

__global__
void wallBoundaryY2nd_U_kernel(
    ConservationParameter* U, 
    int localSizeX, int localSizeY, 
    MPIInfo* device_mPIInfo
)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    MPIInfo mPIInfo = *device_mPIInfo;

    if (i < localSizeX) {
        if (mPIInfo.localGridY == 0) {
            int index = 0 + i * localSizeY;

            double rho, u, v, w, bX, bY, bZ, p, e;
            ConservationParameter wallU;

            for (int buf = 0; buf < mPIInfo.buffer; buf++) {            

                rho = U[index + 2 * mPIInfo.buffer - 1 - buf].rho;
                u   = U[index + 2 * mPIInfo.buffer - 1 - buf].rhoU / rho; 
                v   = U[index + 2 * mPIInfo.buffer - 1 - buf].rhoV / rho; 
                w   = U[index + 2 * mPIInfo.buffer - 1 - buf].rhoW / rho;
                bX  = U[index + 2 * mPIInfo.buffer - 1 - buf].bX; 
                bY  = U[index + 2 * mPIInfo.buffer - 1 - buf].bY;
                bZ  = U[index + 2 * mPIInfo.buffer - 1 - buf].bZ;
                e   = U[index + 2 * mPIInfo.buffer - 1 - buf].e;
                p   = (device_gamma_mhd - 1.0)
                    * (e - 0.5 * rho * (u * u + v * v + w * w)
                    - 0.5 * (bX * bX + bY * bY + bZ * bZ));
                
                wallU.rho = rho;
                wallU.rhoU = rho * u; wallU.rhoV = rho * (-v); wallU.rhoW = rho * w;
                wallU.bX = bX; wallU.bY = 0.0; wallU.bZ = bZ;
                e = p / (device_gamma_mhd - 1.0) + 0.5 * rho * (u * u + (-v) * (-v) + w * w)
                + 0.5 * (bX * bX + 0.0 * 0.0 + bZ * bZ); 
                wallU.e = e;

                U[index + buf] = wallU;
            }
        }
        
        if (mPIInfo.localGridY == mPIInfo.gridY - 1) {
            int index = localSizeY - 1 + i * localSizeY;

            double rho, u, v, w, bX, bY, bZ, p, e;
            ConservationParameter wallU;

            for (int buf = 0; buf < mPIInfo.buffer; buf++) {            

                rho = U[index - 2 * mPIInfo.buffer + 1 + buf].rho;
                u   = U[index - 2 * mPIInfo.buffer + 1 + buf].rhoU / rho; 
                v   = U[index - 2 * mPIInfo.buffer + 1 + buf].rhoV / rho; 
                w   = U[index - 2 * mPIInfo.buffer + 1 + buf].rhoW / rho;
                bX  = U[index - 2 * mPIInfo.buffer + 1 + buf].bX; 
                bY  = U[index - 2 * mPIInfo.buffer + 1 + buf].bY;
                bZ  = U[index - 2 * mPIInfo.buffer + 1 + buf].bZ;
                e   = U[index - 2 * mPIInfo.buffer + 1 + buf].e;
                p   = (device_gamma_mhd - 1.0)
                    * (e - 0.5 * rho * (u * u + v * v + w * w)
                    - 0.5 * (bX * bX + bY * bY + bZ * bZ));
                
                wallU.rho = rho;
                wallU.rhoU = rho * u; wallU.rhoV = rho * (-v); wallU.rhoW = rho * w;
                wallU.bX = bX; wallU.bY = 0.0; wallU.bZ = bZ;
                e = p / (device_gamma_mhd - 1.0) + 0.5 * rho * (u * u + (-v) * (-v) + w * w)
                + 0.5 * (bX * bX + 0.0 * 0.0 + bZ * bZ); 
                wallU.e = e;

                U[index - buf] = wallU;
            }
        }
    }
}

void Boundary::wallBoundaryY2nd_U(
    thrust::device_vector<ConservationParameter>& U
)
{
    // そこまで重くないので、初期化と同じくグローバルで扱うことにする
    int threadsPerBlock = 256;
    int blocksPerGrid = (mPIInfo.localSizeX + threadsPerBlock - 1) / threadsPerBlock;

    wallBoundaryY2nd_U_kernel<<<blocksPerGrid, threadsPerBlock>>>(
        thrust::raw_pointer_cast(U.data()), 
        mPIInfo.localSizeX, mPIInfo.localSizeY, 
        device_mPIInfo
    );
    hipDeviceSynchronize();
}


__global__
void wallBoundaryY2nd_flux_kernel(
    Flux* fluxF, Flux* fluxG, 
    int localSizeX, int localSizeY, 
    MPIInfo* device_mPIInfo
)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    MPIInfo mPIInfo = *device_mPIInfo;

    if (i < localSizeX) {
        if (mPIInfo.localGridY == 0) {
            int index = 0 + i * localSizeY;

            for (int buf = 0; buf < mPIInfo.buffer; buf++) {            
                fluxF[index + buf] = fluxF[index + 2 * mPIInfo.buffer - 1 - buf];
                fluxG[index + buf] = fluxG[index + 2 * mPIInfo.buffer - 1 - buf];
            }
        }
        
        if (mPIInfo.localGridY == mPIInfo.gridY - 1) {
            int index = localSizeY - 1 + i * localSizeY;

            for (int buf = 0; buf < mPIInfo.buffer; buf++) {            

                fluxF[index - buf] = fluxF[index - 2 * mPIInfo.buffer + 1 + buf]; 
                fluxG[index - buf] = fluxG[index - 2 * mPIInfo.buffer + 1 + buf]; 
            }
        }
    }
}

void Boundary::wallBoundaryY2nd_flux(
    thrust::device_vector<Flux>& fluxF, 
    thrust::device_vector<Flux>& fluxG
)
{
    // そこまで重くないので、初期化と同じくグローバルで扱うことにする
    int threadsPerBlock = 256;
    int blocksPerGrid = (mPIInfo.localSizeX + threadsPerBlock - 1) / threadsPerBlock;

    wallBoundaryY2nd_flux_kernel<<<blocksPerGrid, threadsPerBlock>>>(
        thrust::raw_pointer_cast(fluxF.data()), 
        thrust::raw_pointer_cast(fluxG.data()), 
        mPIInfo.localSizeX, mPIInfo.localSizeY, 
        device_mPIInfo
    );
    hipDeviceSynchronize();
}

//////////

__global__
void symmetricBoundaryY2nd_U_kernel(
    ConservationParameter* U, 
    int localSizeX, int localSizeY, 
    MPIInfo* device_mPIInfo
)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    MPIInfo mPIInfo = *device_mPIInfo;

    if (i < localSizeX) {
        if (mPIInfo.localGridY == 0) {
            int index = 0 + i * localSizeY;

            for (int buf = 0; buf < mPIInfo.buffer; buf++) {
                U[index + buf] = U[index + mPIInfo.buffer];
            }
        }
        
        if (mPIInfo.localGridY == mPIInfo.gridY - 1) {
            int index = localSizeY - 1 + i * localSizeY;

            for (int buf = 0; buf < mPIInfo.buffer; buf++) {
                U[index - buf] = U[index - mPIInfo.buffer];
            }
        }
    }
}

void Boundary::symmetricBoundaryY2nd_U(
    thrust::device_vector<ConservationParameter>& U
)
{
    // そこまで重くないので、初期化と同じくグローバルで扱うことにする
    int threadsPerBlock = 256;
    int blocksPerGrid = (mPIInfo.localSizeX + threadsPerBlock - 1) / threadsPerBlock;

    symmetricBoundaryY2nd_U_kernel<<<blocksPerGrid, threadsPerBlock>>>(
        thrust::raw_pointer_cast(U.data()), 
        mPIInfo.localSizeX, mPIInfo.localSizeY, 
        device_mPIInfo
    );
    hipDeviceSynchronize();
}


void Boundary::symmetricBoundaryY2nd_flux(
    thrust::device_vector<Flux>& fluxF, 
    thrust::device_vector<Flux>& fluxG
)
{

}

