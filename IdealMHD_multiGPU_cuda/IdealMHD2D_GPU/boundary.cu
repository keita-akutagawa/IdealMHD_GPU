#include "hip/hip_runtime.h"
#include "boundary.hpp"


Boundary::Boundary(MPIInfo& mPIInfo)
    : mPIInfo(mPIInfo)
{

    hipMalloc(&device_mPIInfo, sizeof(MPIInfo));
    hipMemcpy(device_mPIInfo, &mPIInfo, sizeof(MPIInfo), hipMemcpyHostToDevice);
    
}


void Boundary::periodicBoundaryX2nd_U(
    thrust::device_vector<ConservationParameter>& U
)
{

}


void Boundary::periodicBoundaryY2nd_U(
    thrust::device_vector<ConservationParameter>& U
)
{
    
}

void Boundary::periodicBoundaryX2nd_flux(
    thrust::device_vector<Flux>& fluxF, 
    thrust::device_vector<Flux>& fluxG
)
{

}


void Boundary::periodicBoundaryY2nd_flux(
    thrust::device_vector<Flux>& fluxF, 
    thrust::device_vector<Flux>& fluxG
)
{
    
}

///////////////////////

__global__
void wallBoundaryY2nd_U_kernel(
    ConservationParameter* U, 
    int localSizeX, int localSizeY, 
    MPIInfo* device_mPIInfo
)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    MPIInfo mPIInfo = *device_mPIInfo;

    if (i < localSizeX) {
        if (mPIInfo.localGridY == 0) {
            int index = 0 + i * localSizeY;

            double rho, u, v, w, bX, bY, bZ, p, e;
            ConservationParameter wallU;

            rho = U[index + mPIInfo.buffer].rho;
            u   = U[index + mPIInfo.buffer].rhoU / rho; 
            v   = U[index + mPIInfo.buffer].rhoV / rho; 
            w   = U[index + mPIInfo.buffer].rhoW / rho;
            bX  = U[index + mPIInfo.buffer].bX; 
            bY  = U[index + mPIInfo.buffer].bY;
            bZ  = U[index + mPIInfo.buffer].bZ;
            e   = U[index + mPIInfo.buffer].e;
            p   = (device_gamma_mhd - 1.0)
                * (e - 0.5 * rho * (u * u + v * v + w * w)
                - 0.5 * (bX * bX + bY * bY + bZ * bZ));
            
            wallU.rho = rho;
            wallU.rhoU = rho * 0.0; wallU.rhoV = rho * 0.0; wallU.rhoW = rho * 0.0;
            wallU.bX = bX; wallU.bY = 0.0; wallU.bZ = bZ;
            e = p / (device_gamma_mhd - 1.0) + 0.5 * rho * (0.0 * 0.0 + 0.0 * 0.0 + 0.0 * 0.0)
            + 0.5 * (bX * bX + 0.0 * 0.0 + bZ * bZ); 
            wallU.e = e;

            for (int buf = 0; buf < mPIInfo.buffer; buf++) {            
                U[index + buf] = wallU;
            }
        }
        
        if (mPIInfo.localGridY == mPIInfo.gridY - 1) {
            int index = localSizeY - 1 + i * localSizeY;

            double rho, u, v, w, bX, bY, bZ, p, e;
            ConservationParameter wallU;

            rho = U[index - mPIInfo.buffer].rho;
            u   = U[index - mPIInfo.buffer].rhoU / rho; 
            v   = U[index - mPIInfo.buffer].rhoV / rho; 
            w   = U[index - mPIInfo.buffer].rhoW / rho;
            bX  = U[index - mPIInfo.buffer].bX; 
            bY  = U[index - mPIInfo.buffer].bY;
            bZ  = U[index - mPIInfo.buffer].bZ;
            e   = U[index - mPIInfo.buffer].e;
            p   = (device_gamma_mhd - 1.0)
                * (e - 0.5 * rho * (u * u + v * v + w * w)
                - 0.5 * (bX * bX + bY * bY + bZ * bZ));
            
            wallU.rho = rho;
            wallU.rhoU = rho * 0.0; wallU.rhoV = rho * 0.0; wallU.rhoW = rho * 0.0;
            wallU.bX = bX; wallU.bY = 0.0; wallU.bZ = bZ;
            e = p / (device_gamma_mhd - 1.0) + 0.5 * rho * (0.0 * 0.0 + 0.0 * 0.0 + 0.0 * 0.0)
            + 0.5 * (bX * bX + 0.0 * 0.0 + bZ * bZ); 
            wallU.e = e;

            for (int buf = 0; buf < mPIInfo.buffer; buf++) {
                U[index - buf] = wallU;
            }
        }
    }
}

void Boundary::wallBoundaryY2nd_U(
    thrust::device_vector<ConservationParameter>& U
)
{
    // そこまで重くないので、初期化と同じくグローバルで扱うことにする
    int threadsPerBlock = 256;
    int blocksPerGrid = (mPIInfo.localSizeX + threadsPerBlock - 1) / threadsPerBlock;

    wallBoundaryY2nd_U_kernel<<<blocksPerGrid, threadsPerBlock>>>(
        thrust::raw_pointer_cast(U.data()), 
        mPIInfo.localSizeX, mPIInfo.localSizeY, 
        device_mPIInfo
    );
    hipDeviceSynchronize();
}


// とりあえずCTに使う部分だけ。
__global__
void wallBoundaryY2nd_flux_kernel(
    Flux* fluxF, Flux* fluxG, 
    int localSizeX, int localSizeY, 
    MPIInfo* device_mPIInfo
)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    MPIInfo mPIInfo = *device_mPIInfo;

    if (i < localSizeX) {
        if (mPIInfo.localGridY == 0) {
            int index = 0 + i * localSizeY;

            for (int buf = 0; buf < mPIInfo.buffer; buf++) {  
                fluxF[index + buf] = fluxF[index + mPIInfo.buffer];
                fluxG[index + buf] = fluxG[index + mPIInfo.buffer];

                fluxF[index + buf].f0 = 0.0;
                fluxG[index + buf].f0 = 0.0;
            }
        }
        
        if (mPIInfo.localGridY == mPIInfo.gridY - 1) {
            int index = localSizeY - 1 + i * localSizeY;

            for (int buf = 0; buf < mPIInfo.buffer; buf++) {            
                fluxF[index - buf] = fluxF[index - mPIInfo.buffer];
                fluxG[index - buf] = fluxG[index - mPIInfo.buffer];

                fluxF[index - buf].f0 = 0.0;
                fluxG[index - buf].f0 = 0.0;
            }
        }
    }
}

void Boundary::wallBoundaryY2nd_flux(
    thrust::device_vector<Flux>& fluxF, 
    thrust::device_vector<Flux>& fluxG
)
{
    int threadsPerBlock = 256;
    int blocksPerGrid = (mPIInfo.localSizeX + threadsPerBlock - 1) / threadsPerBlock;

    wallBoundaryY2nd_flux_kernel<<<blocksPerGrid, threadsPerBlock>>>(
        thrust::raw_pointer_cast(fluxF.data()), 
        thrust::raw_pointer_cast(fluxG.data()), 
        mPIInfo.localSizeX, mPIInfo.localSizeY, 
        device_mPIInfo
    );
    hipDeviceSynchronize();
}

//////////

__global__
void symmetricBoundaryY2nd_U_kernel(
    ConservationParameter* U, 
    int localSizeX, int localSizeY, 
    MPIInfo* device_mPIInfo
)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    MPIInfo mPIInfo = *device_mPIInfo;

    if (i < localSizeX) {
        if (mPIInfo.localGridY == 0) {
            int index = 0 + i * localSizeY;

            for (int buf = 0; buf < mPIInfo.buffer; buf++) {
                U[index + buf] = U[index + mPIInfo.buffer];
            }
        }
        
        if (mPIInfo.localGridY == mPIInfo.gridY - 1) {
            int index = localSizeY - 1 + i * localSizeY;

            for (int buf = 0; buf < mPIInfo.buffer; buf++) {
                U[index - buf] = U[index - mPIInfo.buffer];
            }
        }
    }
}

void Boundary::symmetricBoundaryY2nd_U(
    thrust::device_vector<ConservationParameter>& U
)
{
    int threadsPerBlock = 256;
    int blocksPerGrid = (mPIInfo.localSizeX + threadsPerBlock - 1) / threadsPerBlock;

    symmetricBoundaryY2nd_U_kernel<<<blocksPerGrid, threadsPerBlock>>>(
        thrust::raw_pointer_cast(U.data()), 
        mPIInfo.localSizeX, mPIInfo.localSizeY, 
        device_mPIInfo
    );
    hipDeviceSynchronize();
}


__global__
void symmetricBoundaryY2nd_flux_kernel(
    Flux* fluxF, Flux* fluxG, 
    int localSizeX, int localSizeY, 
    MPIInfo* device_mPIInfo
)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    MPIInfo mPIInfo = *device_mPIInfo;

    if (i < localSizeX) {
        if (mPIInfo.localGridY == 0) {
            int index = 0 + i * localSizeY;

            for (int buf = 0; buf < mPIInfo.buffer; buf++) {
                fluxF[index + buf] = fluxF[index + mPIInfo.buffer]; 
                fluxG[index + buf] = fluxG[index + mPIInfo.buffer]; 
            }
        }
        
        if (mPIInfo.localGridY == mPIInfo.gridY - 1) {
            int index = localSizeY - 1 + i * localSizeY;

            for (int buf = 0; buf < mPIInfo.buffer; buf++) {
                fluxF[index - buf] = fluxF[index - mPIInfo.buffer]; 
                fluxG[index - buf] = fluxG[index - mPIInfo.buffer]; 
            }
        }
    }
}

void Boundary::symmetricBoundaryY2nd_flux(
    thrust::device_vector<Flux>& fluxF, 
    thrust::device_vector<Flux>& fluxG
)
{
    int threadsPerBlock = 256;
    int blocksPerGrid = (mPIInfo.localSizeX + threadsPerBlock - 1) / threadsPerBlock;

    symmetricBoundaryY2nd_flux_kernel<<<blocksPerGrid, threadsPerBlock>>>(
        thrust::raw_pointer_cast(fluxF.data()), 
        thrust::raw_pointer_cast(fluxG.data()), 
        mPIInfo.localSizeX, mPIInfo.localSizeY, 
        device_mPIInfo
    );
    hipDeviceSynchronize();
}

