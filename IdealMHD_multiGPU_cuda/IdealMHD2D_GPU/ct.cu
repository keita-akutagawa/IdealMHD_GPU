#include "hip/hip_runtime.h"
#include "ct.hpp"


CT::CT(MPIInfo& mPIInfo)
    : mPIInfo(mPIInfo), 

      oldNumericalFluxF_f5(mPIInfo.localSizeX * mPIInfo.localSizeY), 
      oldNumericalFluxG_f4(mPIInfo.localSizeX * mPIInfo.localSizeY), 
      oldFluxF_f5         (mPIInfo.localSizeX * mPIInfo.localSizeY), 
      oldFluxG_f4         (mPIInfo.localSizeX * mPIInfo.localSizeY), 
      oldNumericalFluxF_f0(mPIInfo.localSizeX * mPIInfo.localSizeY), 
      oldNumericalFluxG_f0(mPIInfo.localSizeX * mPIInfo.localSizeY), 

      nowNumericalFluxF_f5(mPIInfo.localSizeX * mPIInfo.localSizeY), 
      nowNumericalFluxG_f4(mPIInfo.localSizeX * mPIInfo.localSizeY), 
      nowFluxF_f5         (mPIInfo.localSizeX * mPIInfo.localSizeY), 
      nowFluxG_f4         (mPIInfo.localSizeX * mPIInfo.localSizeY), 
      nowNumericalFluxF_f0(mPIInfo.localSizeX * mPIInfo.localSizeY), 
      nowNumericalFluxG_f0(mPIInfo.localSizeX * mPIInfo.localSizeY), 

      eZVector            (mPIInfo.localSizeX * mPIInfo.localSizeY)
{
}


__global__ void setFlux_kernel(
    const Flux* fluxF, const Flux* fluxG, 
    const ConservationParameter* U, 
    double* NumericalFluxF_f5, double* NumericalFluxG_f4, 
    double* FluxF_f5, double* FluxG_f4, 
    double* NumericalFluxF_f0, double* NumericalFluxG_f0, 
    int localSizeX, int localSizeY
)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (0 < i && i < localSizeX - 1 && 0 < j && j < localSizeY - 1) {
        int index = j + i * localSizeY;
        double rho, u, v, bX, bY;
        double xPosition = i * device_dx, yPosition = j * device_dy;
        double jZ;
        double eta;

        rho = U[index].rho;
        u   = U[index].rhoU / rho;
        v   = U[index].rhoV / rho;
        bX  = 0.5 * (U[index].bX + U[index - localSizeY].bX);
        bY  = 0.5 * (U[index].bY + U[index - 1].bY);
        jZ = 0.25 * (
            (U[index + localSizeY].bY - U[index].bY) / device_dx - (U[index + 1].bX - U[index].bX) / device_dy //右上
          + (U[index - 1 + localSizeY].bY - U[index - 1].bY) / device_dx - (U[index].bX - U[index - 1].bX) / device_dy //右下
          + (U[index - 1].bY - U[index - 1 - localSizeY].bY) / device_dx - (U[index - localSizeY].bX - U[index - localSizeY - 1].bX) / device_dy //左下
          + (U[index].bY - U[index - localSizeY].bY) / device_dx - (U[index + 1 - localSizeY].bX - U[index - localSizeY].bX) / device_dy //左上
        );
        eta = getEta(xPosition, yPosition);
  
        NumericalFluxF_f5[index] = fluxF[index].f5;
        NumericalFluxG_f4[index] = fluxG[index].f4;
        FluxF_f5[index] = u * bY - v * bX - eta * jZ;
        FluxG_f4[index] = -(u * bY - v * bX - eta * jZ);
        NumericalFluxF_f0[index] = fluxF[index].f0;
        NumericalFluxG_f0[index] = fluxG[index].f0;
    }
}


void CT::setOldFlux2D(
    const thrust::device_vector<Flux>& fluxF, 
    const thrust::device_vector<Flux>& fluxG, 
    const thrust::device_vector<ConservationParameter>& U
)
{
    dim3 threadsPerBlock(16, 16);
    dim3 blocksPerGrid((mPIInfo.localSizeX + threadsPerBlock.x - 1) / threadsPerBlock.x,
                       (mPIInfo.localSizeY + threadsPerBlock.y - 1) / threadsPerBlock.y);

    setFlux_kernel<<<blocksPerGrid, threadsPerBlock>>>(
        thrust::raw_pointer_cast(fluxF.data()), 
        thrust::raw_pointer_cast(fluxG.data()), 
        thrust::raw_pointer_cast(U.data()), 
        thrust::raw_pointer_cast(oldNumericalFluxF_f5.data()), 
        thrust::raw_pointer_cast(oldNumericalFluxG_f4.data()), 
        thrust::raw_pointer_cast(oldFluxF_f5.data()), 
        thrust::raw_pointer_cast(oldFluxG_f4.data()), 
        thrust::raw_pointer_cast(oldNumericalFluxF_f0.data()), 
        thrust::raw_pointer_cast(oldNumericalFluxG_f0.data()), 
        mPIInfo.localSizeX, mPIInfo.localSizeY
    );
    hipDeviceSynchronize();
}


void CT::setNowFlux2D(
    const thrust::device_vector<Flux>& fluxF, 
    const thrust::device_vector<Flux>& fluxG, 
    const thrust::device_vector<ConservationParameter>& U
)
{
    dim3 threadsPerBlock(16, 16);
    dim3 blocksPerGrid((mPIInfo.localSizeX + threadsPerBlock.x - 1) / threadsPerBlock.x,
                       (mPIInfo.localSizeY + threadsPerBlock.y - 1) / threadsPerBlock.y);

    setFlux_kernel<<<blocksPerGrid, threadsPerBlock>>>(
        thrust::raw_pointer_cast(fluxF.data()), 
        thrust::raw_pointer_cast(fluxG.data()), 
        thrust::raw_pointer_cast(U.data()), 
        thrust::raw_pointer_cast(nowNumericalFluxF_f5.data()), 
        thrust::raw_pointer_cast(nowNumericalFluxG_f4.data()), 
        thrust::raw_pointer_cast(nowFluxF_f5.data()), 
        thrust::raw_pointer_cast(nowFluxG_f4.data()), 
        thrust::raw_pointer_cast(nowNumericalFluxF_f0.data()), 
        thrust::raw_pointer_cast(nowNumericalFluxG_f0.data()), 
        mPIInfo.localSizeX, mPIInfo.localSizeY
    );
    hipDeviceSynchronize();
}


__global__ void getEZVector_kernel(
    const double* oldNumericalFluxF_f5, 
    const double* oldNumericalFluxG_f4, 
    const double* oldFluxF_f5, 
    const double* oldFluxG_f4, 
    const double* oldNumericalFluxF_f0, 
    const double* oldNumericalFluxG_f0, 
    const double* nowNumericalFluxF_f5, 
    const double* nowNumericalFluxG_f4, 
    const double* nowFluxF_f5, 
    const double* nowFluxG_f4, 
    const double* nowNumericalFluxF_f0, 
    const double* nowNumericalFluxG_f0, 
    double* eZVector, 
    int localSizeX, int localSizeY
)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i < localSizeX - 1 && j < localSizeY - 1) {
        double eZOld_arithmeticAverage, eZOld_S, eZOld_N, eZOld_W, eZOld_E;
        double eZOld;
        double eZNow_arithmeticAverage, eZNow_S, eZNow_N, eZNow_W, eZNow_E;
        double eZNow;
        int index = j + i * localSizeY;

        eZOld_arithmeticAverage = 0.25 * (
            - oldNumericalFluxF_f5[index] - oldNumericalFluxF_f5[index + 1]
            + oldNumericalFluxG_f4[index] + oldNumericalFluxG_f4[index + localSizeY]
        );

        eZOld_S = (1.0 + sign(oldNumericalFluxF_f0[index])) 
             * (oldNumericalFluxG_f4[index] - oldFluxG_f4[index])
             + (1.0 - sign(oldNumericalFluxF_f0[index])) 
             * (oldNumericalFluxG_f4[index + localSizeY] - oldFluxG_f4[index + localSizeY]);
        eZOld_N = (1.0 + sign(oldNumericalFluxF_f0[index + 1])) 
             * (oldFluxG_f4[index + 1] - oldNumericalFluxG_f4[index])
             + (1.0 - sign(oldNumericalFluxF_f0[index + 1])) 
             * (oldFluxG_f4[index + 1 + localSizeY] - oldNumericalFluxG_f4[index + localSizeY]);
        eZOld_W = -(1.0 + sign(oldNumericalFluxG_f0[index])) 
             * (oldNumericalFluxF_f5[index] - oldFluxF_f5[index])
             -(1.0 - sign(oldNumericalFluxG_f0[index])) 
             * (oldNumericalFluxF_f5[index + 1] - oldFluxF_f5[index + 1]);
        eZOld_E = -(1.0 + sign(oldNumericalFluxG_f0[index + localSizeY])) 
             * (oldFluxF_f5[index + localSizeY] - oldNumericalFluxF_f5[index])
             -(1.0 - sign(oldNumericalFluxG_f0[index + localSizeY])) 
             * (oldFluxF_f5[index + 1 + localSizeY] - oldNumericalFluxF_f5[index + 1]);

        eZOld = eZOld_arithmeticAverage + 0.125 * (eZOld_S - eZOld_N + eZOld_W - eZOld_E);

        eZNow_arithmeticAverage = 0.25 * (
            - nowNumericalFluxF_f5[index] - nowNumericalFluxF_f5[index + 1]
            + nowNumericalFluxG_f4[index] + nowNumericalFluxG_f4[index + localSizeY]
        );

        eZNow_S = (1.0 + sign(nowNumericalFluxF_f0[index])) 
             * (nowNumericalFluxG_f4[index] - nowFluxG_f4[index])
             + (1.0 - sign(nowNumericalFluxF_f0[index])) 
             * (nowNumericalFluxG_f4[index + localSizeY] - nowFluxG_f4[index + localSizeY]);
        eZNow_N = (1.0 + sign(nowNumericalFluxF_f0[index + 1])) 
             * (nowFluxG_f4[index + 1] - nowNumericalFluxG_f4[index])
             + (1.0 - sign(nowNumericalFluxF_f0[index + 1])) 
             * (nowFluxG_f4[index + 1 + localSizeY] - nowNumericalFluxG_f4[index + localSizeY]);
        eZNow_W = -(1.0 + sign(nowNumericalFluxG_f0[index])) 
             * (nowNumericalFluxF_f5[index] - nowFluxF_f5[index])
             -(1.0 - sign(nowNumericalFluxG_f0[index])) 
             * (nowNumericalFluxF_f5[index + 1] - nowFluxF_f5[index + 1]);
        eZNow_E = -(1.0 + sign(nowNumericalFluxG_f0[index + localSizeY])) 
             * (nowFluxF_f5[index + localSizeY] - nowNumericalFluxF_f5[index])
             -(1.0 - sign(nowNumericalFluxG_f0[index + localSizeY])) 
             * (nowFluxF_f5[index + 1 + localSizeY] - nowNumericalFluxF_f5[index + 1]);

        eZNow = eZNow_arithmeticAverage + 0.125 * (eZNow_S - eZNow_N + eZNow_W - eZNow_E);

        eZVector[index] = 0.5 * (eZOld + eZNow);
    }
}


__global__ void CT_kernel(
    const double* bXOld, const double* bYOld, 
    const double* eZVector, 
    ConservationParameter* U, 
    int localSizeX, int localSizeY
)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if ((0 < i) && (i < localSizeX) && (0 < j) && (j < localSizeY)) {
        int index = j + i * localSizeY;

        U[index].bX = bXOld[index]
                    - device_dt / device_dy * (eZVector[index] - eZVector[index - 1]);
        U[index].bY = bYOld[index]
                    + device_dt / device_dx * (eZVector[index] - eZVector[index - localSizeY]);
    }
}


void CT::divBClean(
    const thrust::device_vector<double>& bXOld, 
    const thrust::device_vector<double>& bYOld, 
    thrust::device_vector<ConservationParameter>& U
)
{
    dim3 threadsPerBlock(16, 16);
    dim3 blocksPerGrid((mPIInfo.localSizeX + threadsPerBlock.x - 1) / threadsPerBlock.x,
                       (mPIInfo.localSizeY + threadsPerBlock.y - 1) / threadsPerBlock.y);

    getEZVector_kernel<<<blocksPerGrid, threadsPerBlock>>>(
        thrust::raw_pointer_cast(oldNumericalFluxF_f5.data()), 
        thrust::raw_pointer_cast(oldNumericalFluxG_f4.data()), 
        thrust::raw_pointer_cast(oldFluxF_f5.data()), 
        thrust::raw_pointer_cast(oldFluxG_f4.data()), 
        thrust::raw_pointer_cast(oldNumericalFluxF_f0.data()), 
        thrust::raw_pointer_cast(oldNumericalFluxG_f0.data()), 
        thrust::raw_pointer_cast(nowNumericalFluxF_f5.data()), 
        thrust::raw_pointer_cast(nowNumericalFluxG_f4.data()), 
        thrust::raw_pointer_cast(nowFluxF_f5.data()), 
        thrust::raw_pointer_cast(nowFluxG_f4.data()), 
        thrust::raw_pointer_cast(nowNumericalFluxF_f0.data()), 
        thrust::raw_pointer_cast(nowNumericalFluxG_f0.data()), 
        thrust::raw_pointer_cast(eZVector.data()), 
        mPIInfo.localSizeX, mPIInfo.localSizeY
    );
    hipDeviceSynchronize();

    CT_kernel<<<blocksPerGrid, threadsPerBlock>>>(
        thrust::raw_pointer_cast(bXOld.data()),
        thrust::raw_pointer_cast(bYOld.data()),
        thrust::raw_pointer_cast(eZVector.data()),
        thrust::raw_pointer_cast(U.data()), 
        mPIInfo.localSizeX, mPIInfo.localSizeY
    );
    hipDeviceSynchronize();
}

