#include "hip/hip_runtime.h"
#include <cmath>
#include <iostream>
#include <fstream>
#include <iomanip>
#include <string>
#include <thrust/extrema.h>
#include "const.hpp"
#include "idealMHD2D.hpp"


IdealMHD2D::IdealMHD2D(MPIInfo& mPIInfo)
    : mPIInfo(mPIInfo), 

      fluxSolver(mPIInfo), 

      fluxF    (mPIInfo.localSizeX * mPIInfo.localSizeY),
      fluxG    (mPIInfo.localSizeX * mPIInfo.localSizeY),
      sourceTerm(mPIInfo.localSizeX * mPIInfo.localSizeY),
      U        (mPIInfo.localSizeX * mPIInfo.localSizeY),
      UBar     (mPIInfo.localSizeX * mPIInfo.localSizeY), 
      tmpVector(mPIInfo.localSizeX * mPIInfo.localSizeY),
      bXOld    (mPIInfo.localSizeX * mPIInfo.localSizeY), 
      bYOld    (mPIInfo.localSizeX * mPIInfo.localSizeY), 
      hU       (mPIInfo.localSizeX * mPIInfo.localSizeY), 

      dtVector(mPIInfo.localNx * mPIInfo.localNy), 

      boundary(mPIInfo), 
      ct(mPIInfo)
{

    hipMalloc(&device_mPIInfo, sizeof(MPIInfo));
    hipMemcpy(device_mPIInfo, &mPIInfo, sizeof(MPIInfo), hipMemcpyHostToDevice);
    
}



__global__ void copyBX_kernel(
    double* tmp, 
    const ConservationParameter* U, 
    int localSizeX, int localSizeY
)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i < localSizeX && j < localSizeY) {
        int index = j + i * localSizeY;

        tmp[index] = U[index].bX;
    }
}

__global__ void copyBY_kernel(
    double* tmp, 
    const ConservationParameter* U, 
    int localSizeX, int localSizeY
)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i < localSizeX && j < localSizeY) {
        int index = j + i * localSizeY;

        tmp[index] = U[index].bY;
    }
}


__global__ void oneStepFirst_kernel(
    const ConservationParameter* U, 
    const Flux* fluxF, const Flux* fluxG, 
    const SourceTerm* sourceTerm, 
    ConservationParameter* UBar, 
    int localSizeX, int localSizeY, 
    const int localGridX, const int localNx, const int buffer 
)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if ((0 < i) && (i < localSizeX) && (0 < j) && (j < localSizeY)) {
        int index = j + i * localSizeY;
        double x = localNx * localGridX + (i - buffer) * device_dx + device_xmin; 

        UBar[index].rho  = U[index].rho  
                         - device_dt / device_dx / x * ((x + 0.5 * device_dx) * fluxF[index].f0 - (x - 0.5 * device_dx) * fluxF[index - localSizeY].f0)
                         - device_dt / device_dy / x * (fluxG[index].f0 - fluxG[index - 1].f0)
                         + device_dt * sourceTerm[index].s0;
        UBar[index].rhoU = U[index].rhoU 
                         - device_dt / device_dx / x * ((x + 0.5 * device_dx) * fluxF[index].f1 - (x - 0.5 * device_dx) * fluxF[index - localSizeY].f1)
                         - device_dt / device_dy / x * (fluxG[index].f1 - fluxG[index - 1].f1)
                         + device_dt * sourceTerm[index].s1;
        UBar[index].rhoV = U[index].rhoV
                         - device_dt / device_dx / x * ((x + 0.5 * device_dx) * fluxF[index].f2 - (x - 0.5 * device_dx) * fluxF[index - localSizeY].f2)
                         - device_dt / device_dy / x * (fluxG[index].f2 - fluxG[index - 1].f2)
                         + device_dt * sourceTerm[index].s2;
        UBar[index].rhoW = U[index].rhoW
                         - device_dt / device_dx / x * ((x + 0.5 * device_dx) * fluxF[index].f3 - (x - 0.5 * device_dx) * fluxF[index - localSizeY].f3)
                         - device_dt / device_dy / x * (fluxG[index].f3 - fluxG[index - 1].f3)
                         + device_dt * sourceTerm[index].s3;
        UBar[index].bX   = U[index].bX 
                         - device_dt / device_dx / x * ((x + 0.5 * device_dx) * fluxF[index].f4 - (x - 0.5 * device_dx) * fluxF[index - localSizeY].f4)
                         - device_dt / device_dy / x * (fluxG[index].f4 - fluxG[index - 1].f4)
                         + device_dt * sourceTerm[index].s4;
        UBar[index].bY   = U[index].bY 
                         - device_dt / device_dx / x * ((x + 0.5 * device_dx) * fluxF[index].f5 - (x - 0.5 * device_dx) * fluxF[index - localSizeY].f5)
                         - device_dt / device_dy / x * (fluxG[index].f5 - fluxG[index - 1].f5)
                         + device_dt * sourceTerm[index].s5;
        UBar[index].bZ   = U[index].bZ 
                         - device_dt / device_dx / x * ((x + 0.5 * device_dx) * fluxF[index].f6 - (x - 0.5 * device_dx) * fluxF[index - localSizeY].f6)
                         - device_dt / device_dy / x * (fluxG[index].f6 - fluxG[index - 1].f6)
                         + device_dt * sourceTerm[index].s6;
        UBar[index].e    = U[index].e 
                         - device_dt / device_dx / x * ((x + 0.5 * device_dx) * fluxF[index].f7 - (x - 0.5 * device_dx) * fluxF[index - localSizeY].f7)
                         - device_dt / device_dy / x * (fluxG[index].f7 - fluxG[index - 1].f7)
                         + device_dt * sourceTerm[index].s7;

    }
}


__global__ void oneStepSecond_kernel(
    const ConservationParameter* UBar, 
    const Flux* fluxF, const Flux* fluxG, 
    const SourceTerm* sourceTerm, 
    ConservationParameter* U, 
    int localSizeX, int localSizeY, 
    const int localGridX, const int localNx, const int buffer 
)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if ((0 < i) && (i < localSizeX) && (0 < j) && (j < localSizeY)) {
        int index = j + i * localSizeY;
        double x = localNx * localGridX + (i - buffer) * device_dx + device_xmin; 

        U[index].rho  = 0.5 * (U[index].rho + UBar[index].rho
                      - device_dt / device_dx / x * ((x + 0.5 * device_dx) * fluxF[index].f0 - (x - 0.5 * device_dx) * fluxF[index - localSizeY].f0)
                      - device_dt / device_dy / x * (fluxG[index].f0 - fluxG[index - 1].f0)
                      + device_dt * sourceTerm[index].s0);
        U[index].rhoU = 0.5 * (U[index].rhoU + UBar[index].rhoU
                      - device_dt / device_dx / x * ((x + 0.5 * device_dx) * fluxF[index].f1 - (x - 0.5 * device_dx) * fluxF[index - localSizeY].f1)
                      - device_dt / device_dy / x * (fluxG[index].f1 - fluxG[index - 1].f1)
                      + device_dt * sourceTerm[index].s1);
        U[index].rhoV = 0.5 * (U[index].rhoV + UBar[index].rhoV
                      - device_dt / device_dx / x * ((x + 0.5 * device_dx) * fluxF[index].f2 - (x - 0.5 * device_dx) * fluxF[index - localSizeY].f2)
                      - device_dt / device_dy / x * (fluxG[index].f2 - fluxG[index - 1].f2)
                      + device_dt * sourceTerm[index].s2);
        U[index].rhoW = 0.5 * (U[index].rhoW + UBar[index].rhoW
                      - device_dt / device_dx / x * ((x + 0.5 * device_dx) * fluxF[index].f3 - (x - 0.5 * device_dx) * fluxF[index - localSizeY].f3)
                      - device_dt / device_dy / x * (fluxG[index].f3 - fluxG[index - 1].f3)
                      + device_dt * sourceTerm[index].s3);
        U[index].bX   = 0.5 * (U[index].bX + UBar[index].bX
                      - device_dt / device_dx / x * ((x + 0.5 * device_dx) * fluxF[index].f4 - (x - 0.5 * device_dx) * fluxF[index - localSizeY].f4)
                      - device_dt / device_dy / x * (fluxG[index].f4 - fluxG[index - 1].f4)
                      + device_dt * sourceTerm[index].s4);
        U[index].bY   = 0.5 * (U[index].bY + UBar[index].bY
                      - device_dt / device_dx / x * ((x + 0.5 * device_dx) * fluxF[index].f5 - (x - 0.5 * device_dx) * fluxF[index - localSizeY].f5)
                      - device_dt / device_dy / x * (fluxG[index].f5 - fluxG[index - 1].f5)
                      + device_dt * sourceTerm[index].s5);
        U[index].bZ   = 0.5 * (U[index].bZ + UBar[index].bZ
                      - device_dt / device_dx / x * ((x + 0.5 * device_dx) * fluxF[index].f6 - (x - 0.5 * device_dx) * fluxF[index - localSizeY].f6)
                      - device_dt / device_dy / x * (fluxG[index].f6 - fluxG[index - 1].f6)
                      + device_dt * sourceTerm[index].s6);
        U[index].e    = 0.5 * (U[index].e + UBar[index].e
                      - device_dt / device_dx / x * ((x + 0.5 * device_dx) * fluxF[index].f7 - (x - 0.5 * device_dx) * fluxF[index - localSizeY].f7)
                      - device_dt / device_dy / x * (fluxG[index].f7 - fluxG[index - 1].f7)
                      + device_dt * sourceTerm[index].s7);
    }
}


void IdealMHD2D::oneStepRK2_symmetricXperiodicY()
{
    dim3 threadsPerBlock(16, 16);
    dim3 blocksPerGrid((mPIInfo.localSizeX + threadsPerBlock.x - 1) / threadsPerBlock.x,
                       (mPIInfo.localSizeY + threadsPerBlock.y - 1) / threadsPerBlock.y);


    MPI_Barrier(MPI_COMM_WORLD);
    
    copyBX_kernel<<<blocksPerGrid, threadsPerBlock>>>(
        thrust::raw_pointer_cast(bXOld.data()), 
        thrust::raw_pointer_cast(U.data()), 
        mPIInfo.localSizeX, mPIInfo.localSizeY
    );
    hipDeviceSynchronize();
    copyBY_kernel<<<blocksPerGrid, threadsPerBlock>>>(
        thrust::raw_pointer_cast(bYOld.data()), 
        thrust::raw_pointer_cast(U.data()), 
        mPIInfo.localSizeX, mPIInfo.localSizeY
    );
    hipDeviceSynchronize();

    thrust::copy(U.begin(), U.end(), UBar.begin());

    calculateDt();

    shiftUToCenterForCT(U);
    fluxF = fluxSolver.getFluxF(U);
    fluxG = fluxSolver.getFluxG(U);
    backUToCenterHalfForCT(U);

    ct.setOldFlux2D(fluxF, fluxG);

    calculateSourceTerm();

    oneStepFirst_kernel<<<blocksPerGrid, threadsPerBlock>>>(
        thrust::raw_pointer_cast(U.data()), 
        thrust::raw_pointer_cast(fluxF.data()), 
        thrust::raw_pointer_cast(fluxG.data()), 
        thrust::raw_pointer_cast(sourceTerm.data()), 
        thrust::raw_pointer_cast(UBar.data()), 
        mPIInfo.localSizeX, mPIInfo.localSizeY, 
        mPIInfo.localGridX, mPIInfo.localNx, mPIInfo.buffer
    );
    hipDeviceSynchronize();

    boundary.wallBoundaryXLeft2nd_U(UBar);
    boundary.symmetricBoundaryXRight2nd_U(UBar);
    boundary.periodicBoundaryY2nd_U(UBar);
    
    shiftUToCenterForCT(UBar);
    fluxF = fluxSolver.getFluxF(UBar);
    fluxG = fluxSolver.getFluxG(UBar);
    backUToCenterHalfForCT(UBar);

    calculateSourceTerm();

    oneStepSecond_kernel<<<blocksPerGrid, threadsPerBlock>>>(
        thrust::raw_pointer_cast(UBar.data()), 
        thrust::raw_pointer_cast(fluxF.data()), 
        thrust::raw_pointer_cast(fluxG.data()), 
        thrust::raw_pointer_cast(sourceTerm.data()),
        thrust::raw_pointer_cast(U.data()), 
        mPIInfo.localSizeX, mPIInfo.localSizeY, 
        mPIInfo.localGridX, mPIInfo.localNx, mPIInfo.buffer
    );
    hipDeviceSynchronize();

    boundary.wallBoundaryXLeft2nd_U(U);
    boundary.symmetricBoundaryXRight2nd_U(U);
    boundary.periodicBoundaryY2nd_U(U);

    ct.divBClean(bXOld, bYOld, U);

    boundary.wallBoundaryXLeft2nd_U(U);
    boundary.symmetricBoundaryXRight2nd_U(U);
    boundary.periodicBoundaryY2nd_U(U);
}


__global__ void calculateSourceTerm_kernel(
    const Flux* fluxF, const Flux* fluxG,  
    SourceTerm* sourceTerm, 
    int localSizeX, int localSizeY, 
    const int localGridX, const int localNx, const int buffer 
)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i < localSizeX && 0 < j && j < localSizeY) {
        int index = j + i * localSizeY;
        double x = localNx * localGridX + (i - buffer) * device_dx + device_xmin; 

        sourceTerm[index].s0 = 0.0;
        sourceTerm[index].s1 = 0.5 * (fluxG[index].f2 + fluxG[index - localSizeY].f2) / x;
        sourceTerm[index].s2 = -0.5 * (fluxF[index].f2 - fluxF[index - localSizeY].f2) / x;
        sourceTerm[index].s3 = 0.0;
        sourceTerm[index].s4 = 0.0;
        sourceTerm[index].s5 = -0.5 * (fluxF[index].f5 + fluxF[index - localSizeY].f5) / x;
        sourceTerm[index].s6 = 0.0;
        sourceTerm[index].s7 = 0.0;
    }
}


void IdealMHD2D::calculateSourceTerm()
{
    dim3 threadsPerBlock(16, 16);
    dim3 blocksPerGrid((mPIInfo.localSizeX + threadsPerBlock.x - 1) / threadsPerBlock.x,
                       (mPIInfo.localSizeY + threadsPerBlock.y - 1) / threadsPerBlock.y);

    calculateSourceTerm_kernel<<<blocksPerGrid, threadsPerBlock>>>(
        thrust::raw_pointer_cast(fluxF.data()), 
        thrust::raw_pointer_cast(fluxG.data()), 
        thrust::raw_pointer_cast(sourceTerm.data()), 
        mPIInfo.localSizeX, mPIInfo.localSizeY, 
        mPIInfo.localGridX, mPIInfo.localNx, mPIInfo.buffer
    );
    hipDeviceSynchronize();
}


void IdealMHD2D::save(
    std::string directoryname, 
    std::string filenameWithoutStep, 
    int step
)
{
    hU = U;

    std::string filename;
    filename = directoryname + "/"
             + filenameWithoutStep + "_" + std::to_string(step)
             + "_" + std::to_string(mPIInfo.rank)
             + ".bin";

    std::ofstream ofs(filename, std::ios::binary);
    ofs << std::fixed << std::setprecision(6);

    for (int i = mPIInfo.buffer; i < mPIInfo.localNx + mPIInfo.buffer; i++) {
        for (int j = mPIInfo.buffer; j < mPIInfo.localNy + mPIInfo.buffer; j++) {
            ofs.write(reinterpret_cast<const char*>(&hU[j + i * mPIInfo.localSizeY].rho),  sizeof(double));
            ofs.write(reinterpret_cast<const char*>(&hU[j + i * mPIInfo.localSizeY].rhoU), sizeof(double));
            ofs.write(reinterpret_cast<const char*>(&hU[j + i * mPIInfo.localSizeY].rhoV), sizeof(double));
            ofs.write(reinterpret_cast<const char*>(&hU[j + i * mPIInfo.localSizeY].rhoW), sizeof(double));
            ofs.write(reinterpret_cast<const char*>(&hU[j + i * mPIInfo.localSizeY].bX),   sizeof(double));
            ofs.write(reinterpret_cast<const char*>(&hU[j + i * mPIInfo.localSizeY].bY),   sizeof(double));
            ofs.write(reinterpret_cast<const char*>(&hU[j + i * mPIInfo.localSizeY].bZ),   sizeof(double));
            ofs.write(reinterpret_cast<const char*>(&hU[j + i * mPIInfo.localSizeY].e),    sizeof(double));
        }
    }
}


__global__ void calculateDtVector_kernel(
    const ConservationParameter* U, 
    double* dtVector, 
    int localNx, int localNy, int buffer, 
    int localGridX
)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i < localNx && j < localNy) {
        int localSizeY = localNy + 2 * buffer;
        int indexForU = (j + buffer) + (i + buffer) * localSizeY;
        int indexForDt = j + i * localNy;
        double x = localNx * localGridX + (i - buffer) * device_dx + device_xmin; 

        double rho, u, v, w, bX, bY, bZ, e, p, cs, ca;
        double maxSpeedX, maxSpeedY;

        rho = U[indexForU].rho;
        u   = U[indexForU].rhoU / rho;
        v   = U[indexForU].rhoV / rho;
        w   = U[indexForU].rhoW / rho;
        bX  = U[indexForU].bX;
        bY  = U[indexForU].bY;
        bZ  = U[indexForU].bZ;
        e   = U[indexForU].e;
        p   = (device_gamma_mhd - 1.0)
            * (e - 0.5 * rho * (u * u + v * v + w * w)
            - 0.5 * (bX * bX + bY * bY + bZ * bZ));
        
        cs = sqrt(device_gamma_mhd * p / rho);
        ca = sqrt((bX * bX + bY * bY + bZ * bZ) / rho);

        maxSpeedX = std::abs(u) + sqrt(cs * cs + ca * ca);
        maxSpeedY = std::abs(v) + sqrt(cs * cs + ca * ca);

        dtVector[indexForDt] = 1.0 / (maxSpeedX / device_dx + maxSpeedY / x / device_dy + device_EPS);
    
    }
}


void IdealMHD2D::calculateDt()
{
    // localSizeではないので注意
    dim3 threadsPerBlock(16, 16);
    dim3 blocksPerGrid((mPIInfo.localNx + threadsPerBlock.x - 1) / threadsPerBlock.x,
                       (mPIInfo.localNy + threadsPerBlock.y - 1) / threadsPerBlock.y);

    calculateDtVector_kernel<<<blocksPerGrid, threadsPerBlock>>>(
        thrust::raw_pointer_cast(U.data()), 
        thrust::raw_pointer_cast(dtVector.data()), 
        mPIInfo.localNx, mPIInfo.localNy, mPIInfo.buffer, 
        mPIInfo.localGridX
    );

    thrust::device_vector<double>::iterator dtMin = thrust::min_element(dtVector.begin(), dtVector.end());
    
    dt = (*dtMin) * CFL;
    
    double dtLocal = dt;
    double dtCommon;
    
    MPI_Allreduce(&dtLocal, &dtCommon, 1, MPI_DOUBLE, MPI_MIN, MPI_COMM_WORLD);

    dt = dtCommon;

    hipMemcpyToSymbol(HIP_SYMBOL(device_dt), &dt, sizeof(double));
    hipDeviceSynchronize();
}


struct IsNan
{
    __device__ 
    bool operator()(const ConservationParameter U) const {
        return isnan(U.e); // 何かが壊れたらeは壊れるから
    }
};

bool IdealMHD2D::checkCalculationIsCrashed()
{
    bool result = thrust::transform_reduce(
        U.begin(), U.end(), IsNan(), false, thrust::logical_or<bool>()
    );

    bool global_result;
    MPI_Allreduce(&result, &global_result, 1, MPI_C_BOOL, MPI_LOR, MPI_COMM_WORLD);

    if (dt < 0) global_result = true;

    return global_result;
}

/////////////////////

__global__ void shiftBXToCenterForCT_kernel(
    const double* tmp, 
    ConservationParameter* U, 
    int localSizeX, int localSizeY
)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if ((0 < i) && (i < localSizeX) && (j < localSizeY)) {
        int index = j + i * localSizeY;

        U[index].bX = 0.5 * (tmp[index] + tmp[index - localSizeY]);
    }
}

__global__ void shiftBYToCenterForCT_kernel(
    const double* tmp, 
    ConservationParameter* U, 
    int localSizeX, int localSizeY
)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if ((i < localSizeX) && (0 < j) && (j < localSizeY)) {
        int index = j + i * localSizeY;

        U[index].bY = 0.5 * (tmp[index] + tmp[index - 1]);
    }
}


void IdealMHD2D::shiftUToCenterForCT(
    thrust::device_vector<ConservationParameter>& U
)
{
    dim3 threadsPerBlock(16, 16);
    dim3 blocksPerGrid((mPIInfo.localSizeX + threadsPerBlock.x - 1) / threadsPerBlock.x,
                       (mPIInfo.localSizeY + threadsPerBlock.y - 1) / threadsPerBlock.y);


    copyBX_kernel<<<blocksPerGrid, threadsPerBlock>>>(
        thrust::raw_pointer_cast(tmpVector.data()), 
        thrust::raw_pointer_cast(U.data()), 
        mPIInfo.localSizeX, mPIInfo.localSizeY
    );
    
    shiftBXToCenterForCT_kernel<<<blocksPerGrid, threadsPerBlock>>>(
        thrust::raw_pointer_cast(tmpVector.data()), 
        thrust::raw_pointer_cast(U.data()), 
        mPIInfo.localSizeX, mPIInfo.localSizeY
    );

    copyBY_kernel<<<blocksPerGrid, threadsPerBlock>>>(
        thrust::raw_pointer_cast(tmpVector.data()), 
        thrust::raw_pointer_cast(U.data()), 
        mPIInfo.localSizeX, mPIInfo.localSizeY
    );
    
    shiftBYToCenterForCT_kernel<<<blocksPerGrid, threadsPerBlock>>>(
        thrust::raw_pointer_cast(tmpVector.data()), 
        thrust::raw_pointer_cast(U.data()), 
        mPIInfo.localSizeX, mPIInfo.localSizeY
    );
}


__global__ void backBXToCenterForCT_kernel(
    const double* tmp, 
    ConservationParameter* U, 
    int localSizeX, int localSizeY
)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if ((i < localSizeX - 1) && (j < localSizeY)) {
        int index = j + i * localSizeY;

        U[index].bX = 0.5 * (tmp[index] + tmp[index + localSizeY]);
    }
}

__global__ void backBYToCenterForCT_kernel(
    const double* tmp, 
    ConservationParameter* U, 
    int localSizeX, int localSizeY
)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if ((i < localSizeX) && (j < localSizeY - 1)) {
        int index = j + i * localSizeY;

        U[index].bY = 0.5 * (tmp[index] + tmp[index + 1]);
    }
}


void IdealMHD2D::backUToCenterHalfForCT(
    thrust::device_vector<ConservationParameter>& U
)
{
    dim3 threadsPerBlock(16, 16);
    dim3 blocksPerGrid((mPIInfo.localSizeX + threadsPerBlock.x - 1) / threadsPerBlock.x,
                       (mPIInfo.localSizeY + threadsPerBlock.y - 1) / threadsPerBlock.y);


    copyBX_kernel<<<blocksPerGrid, threadsPerBlock>>>(
        thrust::raw_pointer_cast(tmpVector.data()), 
        thrust::raw_pointer_cast(U.data()), 
        mPIInfo.localSizeX, mPIInfo.localSizeY
    );
    
    backBXToCenterForCT_kernel<<<blocksPerGrid, threadsPerBlock>>>(
        thrust::raw_pointer_cast(tmpVector.data()), 
        thrust::raw_pointer_cast(U.data()), 
        mPIInfo.localSizeX, mPIInfo.localSizeY
    );

    copyBY_kernel<<<blocksPerGrid, threadsPerBlock>>>(
        thrust::raw_pointer_cast(tmpVector.data()), 
        thrust::raw_pointer_cast(U.data()), 
        mPIInfo.localSizeX, mPIInfo.localSizeY
    );
    
    backBYToCenterForCT_kernel<<<blocksPerGrid, threadsPerBlock>>>(
        thrust::raw_pointer_cast(tmpVector.data()), 
        thrust::raw_pointer_cast(U.data()), 
        mPIInfo.localSizeX, mPIInfo.localSizeY
    );
}



thrust::device_vector<ConservationParameter>& IdealMHD2D::getU()
{
    return U;
}

