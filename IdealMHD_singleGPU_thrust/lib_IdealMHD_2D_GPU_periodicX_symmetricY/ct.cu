#include "hip/hip_runtime.h"
#include "ct.hpp"


CT::CT()
    : oldFluxF(nx * ny), 
      oldFluxG(nx * ny), 
      eZVector(nx * ny)
{
}


void CT::setOldFlux2D(
    const thrust::device_vector<Flux>& fluxF, 
    const thrust::device_vector<Flux>& fluxG
)
{
    thrust::copy(fluxF.begin(), fluxF.end(), oldFluxF.begin());
    thrust::copy(fluxG.begin(), fluxG.end(), oldFluxG.begin());
}


__global__ void getEZVector_kernel(
    const Flux* fluxF, 
    const Flux* fluxG, 
    double* eZVector
)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i < device_nx - 1 && j < device_ny - 1) {
        double eZF1, eZF2, eZG1, eZG2, eZ;

        eZG1 = fluxG[j + i * device_ny].f4;
        eZG2 = fluxG[j + (i + 1) * device_ny].f4;
        eZF1 = -1.0 * fluxF[j + i * device_ny].f5;
        eZF2 = -1.0 * fluxF[j + 1 + i * device_ny].f5;
        eZ = 0.25 * (eZG1 + eZG2 + eZF1 + eZF2);
        eZVector[j + i * device_ny] = eZ;
    }
}


__global__ void CT_kernel(
    const double* bXOld, const double* bYOld, 
    const double* eZVector, 
    ConservationParameter* U
)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if ((0 < i) && (i < device_nx) && (0 < j) && (j < device_ny)) {
        U[j + i * device_ny].bX = bXOld[j + i * device_ny]
                                - device_dt / device_dy * (eZVector[j + i * device_ny] - eZVector[j - 1 + i * device_ny]);
        U[j + i * device_ny].bY = bYOld[j + i * device_ny]
                                + device_dt / device_dx * (eZVector[j + i * device_ny] - eZVector[j + (i - 1) * device_ny]);
    }
}


void CT::divBClean(
    const thrust::device_vector<double>& bXOld, 
    const thrust::device_vector<double>& bYOld, 
    thrust::device_vector<ConservationParameter>& U
)
{
    dim3 threadsPerBlock(16, 16);
    dim3 blocksPerGrid((nx + threadsPerBlock.x - 1) / threadsPerBlock.x,
                       (ny + threadsPerBlock.y - 1) / threadsPerBlock.y);

    getEZVector_kernel<<<blocksPerGrid, threadsPerBlock>>>(
        thrust::raw_pointer_cast(oldFluxF.data()), 
        thrust::raw_pointer_cast(oldFluxG.data()), 
        thrust::raw_pointer_cast(eZVector.data())
    );

    CT_kernel<<<blocksPerGrid, threadsPerBlock>>>(
        thrust::raw_pointer_cast(bXOld.data()),
        thrust::raw_pointer_cast(bYOld.data()),
        thrust::raw_pointer_cast(eZVector.data()),
        thrust::raw_pointer_cast(U.data())
    );

}

